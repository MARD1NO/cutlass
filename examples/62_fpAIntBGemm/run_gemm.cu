#include <cutlass/core_io.h>
#include <cutlass/cutlass.h>
#include <cutlass/half.h>

#include <cutlass/gemm/device/gemm.h>
#include <cutlass/numeric_types.h>
#include <cutlass/util/host_tensor.h>
#include "cutlass/util/command_line.h"
#include "stdio.h"
#include "fpA_intB_gemm_template.h"


int main(int argc, const char** args){
    int M = 0;
    int N = 0;
    int K = 0;
    cutlass::CommandLine cmd(argc, args);

    cmd.get_cmd_line_argument("M", M, 1);
    cmd.get_cmd_line_argument("N", N, 1);
    cmd.get_cmd_line_argument("K", K, 1);
    
    half* input; 
    uint8_t* weight; 
    half* out;
    half* weight_scale; 

    hipMalloc(&input, sizeof(half) * M * K); 
    hipMalloc(&weight, sizeof(uint8_t) * K * N); 
    hipMalloc(&out, sizeof(half) * M * N); 
    hipMalloc(&weight_scale, sizeof(half) * K * N); 

    hipStream_t stream; 
    hipStreamCreate(&stream); 

    fastertransformer::CutlassFpAIntBGemmRunner<half, uint8_t> runner{}; 
    char* mixed_gemm_workspace_; 
    int mixed_gemm_ws_bytes_  = runner.getWorkspaceSize(M, N, K);
    hipMalloc(&mixed_gemm_workspace_, mixed_gemm_ws_bytes_); 
    // mixed_gemm_workspace_ = (char*)allocator_->reMalloc(mixed_gemm_workspace_, mixed_gemm_ws_bytes_, false);

    runner.gemm(
        // reinterpret_cast<const cutlass::half_t*>(input), 
        input, 
        weight, 
        // reinterpret_cast<const cutlass::half_t*>(weight_scale), 
        // reinterpret_cast<cutlass::half_t*>(out), 
        weight_scale, 
        out, 
        M, 
        N, 
        K, 
        mixed_gemm_workspace_, 
        mixed_gemm_ws_bytes_, 
        stream
    ); 

    hipFree(input); 
    hipFree(weight); 
    hipFree(out); 
    hipFree(weight_scale); 
    hipStreamDestroy(stream); 
    hipFree(mixed_gemm_workspace_); 
}  
