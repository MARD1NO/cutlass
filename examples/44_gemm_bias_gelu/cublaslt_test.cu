
#include <hip/hip_runtime.h>
#include "iostream"

#include <hipblaslt.h>
#include <hipblas.h>


// Usage: nvcc -O3 cublaslt_test.cu -lcublasLt -o cublaslt_test

void SetCublasMatrixLayout(hipblasLtMatrixLayout_t layout_desc,
                           hipblasOperation_t cublas_trans,
                           const size_t cublas_m,
                           const size_t cublas_n) {
    hipblasLtMatrixLayoutSetAttribute(
            layout_desc,
            HIPBLASLT_MATRIX_LAYOUT_ROWS,
            cublas_trans == HIPBLAS_OP_N ? &cublas_m : &cublas_n,
            sizeof(cublas_m));
    hipblasLtMatrixLayoutSetAttribute(
            layout_desc,
            HIPBLASLT_MATRIX_LAYOUT_COLS,
            cublas_trans == HIPBLAS_OP_N ? &cublas_n : &cublas_m,
            sizeof(cublas_m));
    const size_t cublas_ld = cublas_trans == HIPBLAS_OP_N ? cublas_m : cublas_n;
    hipblasLtMatrixLayoutSetAttribute(
            layout_desc,
            HIPBLASLT_MATRIX_LAYOUT_LD,
            &cublas_ld,
            sizeof(cublas_ld));
  }

struct Result {

  double runtime_ms;
  hipError_t error;
  //
  // Methods
  //
  Result(
    double runtime_ms = 0,
    hipError_t error = hipSuccess
  ): runtime_ms(runtime_ms), error(error){ }
};

int main(){
    hipDataType mat_type = HIP_R_16F;
    hipDataType scale_type = HIP_R_32F;
    hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_32F;

    hipblasLtMatmulDesc_t operation_desc_;
    hipblasLtMatrixLayout_t x_desc_;
    hipblasLtMatrixLayout_t w_desc_;
    hipblasLtMatrixLayout_t out_desc_;

    hipblasLtMatmulDescCreate(&operation_desc_, compute_type, scale_type);
    hipblasLtMatrixLayoutCreate(&x_desc_, mat_type, 1, 1, 1);
    hipblasLtMatrixLayoutCreate(&w_desc_, mat_type, 1, 1, 1);
    hipblasLtMatrixLayoutCreate(&out_desc_, mat_type, 1, 1, 1);

    // int64_t bsz_seq = 2048; 
    // int64_t in_feature = 16384; 
    // int64_t hidden_feature = 4096; 

    int64_t bsz_seq = 5120; 
    int64_t in_feature = 4096; 
    int64_t hidden_feature = 4096; 

    // int64_t bsz_seq = 4096; 
    // int64_t in_feature = 4096; 
    // int64_t hidden_feature = 5120; 

    int64_t M = bsz_seq;
    int64_t K = in_feature;
    int64_t N = hidden_feature;

    half* w_data; 
    half* x_data; 
    half* out_data; 
    half* bias_data; 
    uint8_t* workspace; 
    size_t workspace_size = 16 * 1024 * 1024; 
    hipMalloc(&x_data, sizeof(half)* M * K); 
    hipMalloc(&w_data, sizeof(half)* K * N); 
    hipMalloc(&out_data, sizeof(half)* M * N); 
    hipMalloc(&bias_data, sizeof(half)* N); 
    hipMalloc(&workspace, workspace_size); 

    hipblasOperation_t cublas_transA = HIPBLAS_OP_N;
    hipblasOperation_t cublas_transB = HIPBLAS_OP_N;
    hipblasLtMatmulDescSetAttribute(
            operation_desc_,
            HIPBLASLT_MATMUL_DESC_TRANSB,
            &cublas_transA,
            sizeof(cublas_transA));
    hipblasLtMatmulDescSetAttribute(
            operation_desc_,
            HIPBLASLT_MATMUL_DESC_TRANSA,
            &cublas_transB,
            sizeof(cublas_transB));

    hipblasLtEpilogue_t epiloque_func = HIPBLASLT_EPILOGUE_GELU_BIAS;
    // cublasLtEpilogue_t epiloque_func = CUBLASLT_EPILOGUE_RELU_BIAS;

    hipblasLtMatmulDescSetAttribute(
            operation_desc_,
            HIPBLASLT_MATMUL_DESC_EPILOGUE,
            &epiloque_func,
            sizeof(epiloque_func));

    hipblasLtMatmulDescSetAttribute(
        operation_desc_,
        HIPBLASLT_MATMUL_DESC_BIAS_POINTER,
        &bias_data,
        sizeof(bias_data));

    /*
    cublas use col major: x(M, K) matmul w(K, N) = out(M, N) equals to w_t(N, K)
    * x_t(K, M) = out(N, M)
    */
    SetCublasMatrixLayout(x_desc_, cublas_transA, K, M);
    SetCublasMatrixLayout(w_desc_, cublas_transB, N, K);
    SetCublasMatrixLayout(out_desc_, HIPBLAS_OP_N, N, M);

    hipStream_t stream; 
    hipStreamCreate(&stream); 

    hipblasLtHandle_t handle; 
    hipblasLtCreate(&handle); 

    float alpha32 = 1.0f, beta32 = 0.0f;
    void *alpha = nullptr, *beta = nullptr;
    alpha = &alpha32;
    beta = &beta32;

    hipblasStatus_t cublas_status; 
    
    cublas_status = hipblasLtMatmul(handle,
                                operation_desc_,
                                alpha,
                                w_data,
                                w_desc_,
                                x_data,
                                x_desc_,
                                beta,
                                out_data,
                                out_desc_,
                                out_data,
                                out_desc_,
                                nullptr /*algo*/,
                                workspace /*workspace*/,
                                workspace_size,
                                stream);
    if(cublas_status != HIPBLAS_STATUS_SUCCESS){
        printf("Error. \n"); 
    }
    Result profile_result;

    hipEvent_t events[2];

    for (auto & event : events) {
        profile_result.error = hipEventCreate(&event);
        if (profile_result.error != hipSuccess) {
        std::cerr << "cudaEventCreate() failed: " << hipGetErrorString(profile_result.error) << std::endl;
        return -1;
        }
    }

    // Record an event at the start of a series of GEMM operations
    profile_result.error = hipEventRecord(events[0]);
    if (profile_result.error != hipSuccess) {
        std::cerr << "cudaEventRecord() failed: " << hipGetErrorString(profile_result.error) << std::endl;
        return -1; 
    }

    //
    // Run profiling loop
    //
    const int32_t iter_num = 1; 

    for (int iter = 0; iter < iter_num; ++iter) {
        hipblasLtMatmul(handle,
                        operation_desc_,
                        alpha,
                        w_data,
                        w_desc_,
                        x_data,
                        x_desc_,
                        beta,
                        out_data,
                        out_desc_,
                        out_data,
                        out_desc_,
                        nullptr /*algo*/,
                        workspace /*workspace*/,
                        workspace_size,
                        stream);
        if(cublas_status != HIPBLAS_STATUS_SUCCESS){
            printf("Error. \n"); 
        }
    }

    profile_result.error = hipEventRecord(events[1]);
    if (profile_result.error != hipSuccess) {
        std::cerr << "cudaEventRecord() failed: " << hipGetErrorString(profile_result.error) << std::endl;
        return -1; 
    }

    // Wait for work on the device to complete.
    profile_result.error = hipEventSynchronize(events[1]);
    if (profile_result.error != hipSuccess) {
        std::cerr << "cudaEventSynchronize() failed: " << hipGetErrorString(profile_result.error) << std::endl;
        return -1;
    }

    // Measure elapsed runtime
    float runtime_ms = 0;
    profile_result.error = hipEventElapsedTime(&runtime_ms, events[0], events[1]);
    if (profile_result.error != hipSuccess) {
        std::cerr << "cudaEventElapsed() failed: " << hipGetErrorString(profile_result.error) << std::endl;
        return -1;
    }

    // Compute average runtime and GFLOPs.
    profile_result.runtime_ms = double(runtime_ms) / double(iter_num);
    
    //
    // Cleanup
    //

    for (auto event : events) {
        (void)hipEventDestroy(event);
    }

    hipDeviceSynchronize();

    std::cout << std::endl;
    std::cout << "cublasLt Gemm+Bias+GELU:\n"
        << "====================================================" << std::endl;
    std::cout << "    " << " {M, K, N} = {" << M \
        << ", " << K << ", " << N <<"}." << std::endl;
    std::cout << std::endl;
    std::cout << "    " << "Runtime: " << profile_result.runtime_ms << " ms" << std::endl;

    hipFree(x_data); 
    hipFree(w_data); 
    hipFree(out_data); 
    hipFree(bias_data); 
    hipFree(workspace); 
    hipStreamDestroy(stream); 
    hipblasLtDestroy(handle); 
    return 0;
}   